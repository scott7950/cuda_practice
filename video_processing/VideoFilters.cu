#include "hip/hip_runtime.h"
/******************************************************************************
*
*            (C) Copyright 2014 The Board of Trustees of the
*                        Florida Institute of Technology
*                         All Rights Reserved
*
* Lab Image Filters
******************************************************************************/
#include "VideoFilters.h"

//CUDA 
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <helper_string.h>
#include <hip/hip_texture_types.h>
#include <>

/*TEXTURES*/
texture<uchar4, 2, hipReadModeNormalizedFloat> texImage_rgb;


hipChannelFormatDesc uchar4tex = hipCreateChannelDesc<uchar4>();

hipArray *cu_image;

extern "C"
void CUDA_CreateMemoryArray(int imageW,int imageH){
	hipMallocArray(&cu_image, &uchar4tex, imageW, imageH);
}

extern "C"
void CUDA_BindTextureToArray(){
	hipBindTextureToArray(texImage_rgb,cu_image);
}

extern "C"
void CUDA_FreeArrays(){
	hipFreeArray(cu_image);
}

extern "C"
void CUDA_MemcpyToArray(uchar4 *src,int imageW,int imageH){
	hipMemcpyToArray( cu_image, 0, 0,src, imageW * imageH * sizeof(uchar4), hipMemcpyDeviceToDevice);
}

/***************************************
	Box Filter
*****************************************/

__constant__ float MeanKernel[9] = {1,1,1,  
                                    1,1,1,
									1,1,1};

/***************************************
	Median Filter Kernel Function
*****************************************/
__global__ void MeanFilter(uchar4 *Image_dev, int w, int h){
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    const float x=(float)ix+0.5f;
    const float y=(float)iy+0.5f;

	int win_W=1;

	if(ix < w && iy < h){
		float4 pixval;
		float3 sum;
		sum.x=0.0f;
		sum.y=0.0f;
		sum.z=0.0f;
		int k=0;
		for(int ii=-win_W;ii<=win_W;ii++){
			for(int jj=-win_W;jj<=win_W;jj++){
				pixval=tex2D(texImage_rgb,x+ii,y+jj);
				sum.x+=pixval.x*MeanKernel[k];
				sum.y+=pixval.y*MeanKernel[k];
				sum.z+=pixval.z*MeanKernel[k];
				k++;
			}
		}
		Image_dev[w*iy+ix].x=(unsigned char)((sum.x/9)*255);
		Image_dev[w*iy+ix].y=(unsigned char)((sum.y/9)*255);
		Image_dev[w*iy+ix].z=(unsigned char)((sum.z/9)*255);
	}
}

/***************************************
	Median Filter Calling Function
*****************************************/
extern "C"
void CUDA_MeanFilter(uchar4 *Image_dev,int imageW,int imageH,dim3 grid,dim3 threads){
	MeanFilter<<<grid,threads>>>(Image_dev,imageW,imageH);
}

/***************************************
 *   Gaussian Filter
*****************************************/
__constant__ float GaussianKernel[9] = {1,2,1,  
                                        2,4,2,
									    1,2,1};

__constant__ float GaussianKernel55[25] = {1,4,7,4,1,
										   4,16,26,16,4,
										   7,26,41,26,7,
										   4,16,26,16,4,
										   1,4,7,4,1};

/***************************************
	Gaussian Filter Kernel Function
*****************************************/
__global__ void GaussianFilter(uchar4 *Image_dev, int w, int h){
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const float x = (float)ix + 0.5f;
	const float y = (float)iy + 0.5f;

	//int win_W = 1;
	int win_W = 2;
	//int normalizingFactor = 16;
	int normalizingFactor_55 = 273;

	if (ix < w && iy < h){
		float4 pixval;
		float3 sum;
		sum.x = 0.0f;
		sum.y = 0.0f;
		sum.z = 0.0f;
		int k = 0;
		for (int ii = -win_W; ii <= win_W; ii++){
			for (int jj = -win_W; jj <= win_W; jj++){
				pixval = tex2D(texImage_rgb, x + ii, y + jj);
				//sum.x += pixval.x*GaussianKernel[k];
				//sum.y += pixval.y*GaussianKernel[k];
				//sum.z += pixval.z*GaussianKernel[k];
				sum.x += pixval.x*GaussianKernel55[k];
				sum.y += pixval.y*GaussianKernel55[k];
				sum.z += pixval.z*GaussianKernel55[k];
				k++;
			}
		}
		//Image_dev[w*iy + ix].x = (unsigned char)((sum.x / normalizingFactor) * 255);
		//Image_dev[w*iy + ix].y = (unsigned char)((sum.y / normalizingFactor) * 255);
		//Image_dev[w*iy + ix].z = (unsigned char)((sum.z / normalizingFactor) * 255);
		Image_dev[w*iy + ix].x = (unsigned char)((sum.x / normalizingFactor_55) * 255);
		Image_dev[w*iy + ix].y = (unsigned char)((sum.y / normalizingFactor_55) * 255);
		Image_dev[w*iy + ix].z = (unsigned char)((sum.z / normalizingFactor_55) * 255);
	}
}

/***************************************
	Gaussian Filter Calling Function
*****************************************/
extern "C"
void CUDA_GaussianFilter(uchar4 *Image_dev, int imageW, int imageH, dim3 grid, dim3 threads){
	GaussianFilter << <grid, threads >> >(Image_dev, imageW, imageH);
}


/***************************************
*   Laplacian Filter
*****************************************/

__constant__ float LaplacianKernel[9] = { 0, 1, 0,
										1, -4, 1,
										0, 1, 0 };

/***************************************
	Laplacian Filter Kernel Function
*****************************************/
__global__ void LaplacianFilter(float *Image_dev, int w, int h){
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const float x = (float)ix + 0.5f;
	const float y = (float)iy + 0.5f;

	int win_W = 1;

	if (ix < w && iy < h){
		float4 pixval;
		float sum = 0.0f;

		int k = 0;
		for (int ii = -win_W; ii <= win_W; ii++){
			for (int jj = -win_W; jj <= win_W; jj++){
				pixval = tex2D(texImage_rgb, x + ii, y + jj);
				float Intensidad = (pixval.x + pixval.y + pixval.z) / 3.0f;

				sum += Intensidad*LaplacianKernel[k];

				k++;
			}
		}
		Image_dev[w*iy + ix] = sum;
	}
}

/***************************************
	Laplacian Filter Calling Function
*****************************************/
void CUDA_LaplacianFilter(float *Image_dev, int imageW, int imageH, dim3 grid, dim3 threads){
	LaplacianFilter << <grid, threads >> >(Image_dev, imageW, imageH);
}
