#include "hip/hip_runtime.h"
/******************************************************************************
*
*            (C) Copyright 2014 The Board of Trustees of the
*                        Florida Institute of Technology
*                         All Rights Reserved
*
* Lab Image Filters
******************************************************************************/
#include "ImageFilters.h"

//CUDA 
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <helper_string.h>
#include <hip/hip_texture_types.h>
#include <>

/*TEXTURES*/
texture<uchar4, 2, hipReadModeNormalizedFloat> texImage_rgb;


hipChannelFormatDesc uchar4tex = hipCreateChannelDesc<uchar4>();

hipArray *cu_image;

extern "C"
void CUDA_CreateMemoryArray(int imageW,int imageH){
	hipMallocArray(&cu_image, &uchar4tex, imageW, imageH);
}

extern "C"
void CUDA_BindTextureToArray(){
	hipBindTextureToArray(texImage_rgb,cu_image);
}

extern "C"
void CUDA_FreeArrays(){
	hipFreeArray(cu_image);
}

extern "C"
void CUDA_MemcpyToArray(uchar4 *src,int imageW,int imageH){
	hipMemcpyToArray( cu_image, 0, 0,src, imageW * imageH * sizeof(uchar4), hipMemcpyDeviceToDevice);
}

/***************************************
	Box Filter
*****************************************/

__constant__ float MeanKernel[9] = {1,1,1,  
                                    1,1,1,
									1,1,1};

__constant__ float MeanKernel_55[25] = { 1, 1, 1, 1, 1,
										1, 1, 1, 1, 1,
										1, 1, 1, 1, 1,
										1, 1, 1, 1, 1,
										1, 1, 1, 1, 1 };

__constant__ float MeanKernel_77[49] = { 1, 1, 1, 1, 1, 1, 1,
								         1, 1, 1, 1, 1, 1, 1,
										 1, 1, 1, 1, 1, 1, 1,
										 1, 1, 1, 1, 1, 1, 1,
										 1, 1, 1, 1, 1, 1, 1,
										 1, 1, 1, 1, 1, 1, 1,
										 1, 1, 1, 1, 1, 1, 1 };

/***************************************
	Mean Filter Kernel Function
*****************************************/
__global__ void MeanFilter(uchar4 *Image_dev, int w, int h){
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    const float x=(float)ix+0.5f;
    const float y=(float)iy+0.5f;

	int win_W=1;
	//int win_W = 2;
	//int win_W = 3;

	if(ix < w && iy < h){
		float4 pixval;
		float3 sum;
		sum.x=0.0f;
		sum.y=0.0f;
		sum.z=0.0f;
		int k=0;
		for(int ii=-win_W;ii<=win_W;ii++){
			for(int jj=-win_W;jj<=win_W;jj++){
				pixval=tex2D(texImage_rgb,x+ii,y+jj);

				sum.x += pixval.x*MeanKernel[k];
				sum.y += pixval.y*MeanKernel[k];
				sum.z += pixval.z*MeanKernel[k];

				//sum.x += pixval.x*MeanKernel_55[k];
				//sum.y += pixval.y*MeanKernel_55[k];
				//sum.z += pixval.z*MeanKernel_55[k];

				//sum.x += pixval.x*MeanKernel_77[k];
				//sum.y += pixval.y*MeanKernel_77[k];
				//sum.z += pixval.z*MeanKernel_77[k];
				k++;
			}
		}
		Image_dev[w*iy+ix].x=(unsigned char)((sum.x/9)*255);
		Image_dev[w*iy+ix].y=(unsigned char)((sum.y/9)*255);
		Image_dev[w*iy+ix].z=(unsigned char)((sum.z/9)*255);

		//Image_dev[w*iy + ix].x = (unsigned char)((sum.x / 25) * 255);
		//Image_dev[w*iy + ix].y = (unsigned char)((sum.y / 25) * 255);
		//Image_dev[w*iy + ix].z = (unsigned char)((sum.z / 25) * 255);

		//Image_dev[w*iy + ix].x = (unsigned char)((sum.x / 49) * 255);
		//Image_dev[w*iy + ix].y = (unsigned char)((sum.y / 49) * 255);
		//Image_dev[w*iy + ix].z = (unsigned char)((sum.z / 49) * 255);
	}
}

/***************************************
	Mean Filter Calling Function
*****************************************/
extern "C"
void CUDA_MeanFilter(uchar4 *Image_dev,int imageW,int imageH,dim3 grid,dim3 threads){
	MeanFilter<<<grid,threads>>>(Image_dev,imageW,imageH);
}
